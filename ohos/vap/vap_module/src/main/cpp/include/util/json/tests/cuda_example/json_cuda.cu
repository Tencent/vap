//     __ _____ _____ _____
//  __|  |   __|     |   | |  JSON for Modern C++ (supporting code)
// |  |  |__   |  |  | | | |  version 3.11.3
// |_____|_____|_____|_|___|  https://github.com/nlohmann/json
//
// SPDX-FileCopyrightText: 2013 - 2024 Niels Lohmann <https://nlohmann.me>
// SPDX-License-Identifier: MIT

#include <nlohmann/json.hpp>

int main()
{
    nlohmann::ordered_json json = {"Test"};
    json.dump();

    // regression for #3013 (ordered_json::reset() compile error with nvcc)
    nlohmann::ordered_json metadata;
    metadata.erase("key");
}
